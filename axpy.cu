#include "hip/hip_runtime.h"
#include <iostream>
#include <vector>

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

#define CHECK_CUDA_ERROR(error) CheckCudaError((error), __FILE__, __LINE__)

inline void CheckCudaError(
    hipError_t error,
    const char* file,
    int line,
    bool abort = true) {
  if (error != hipSuccess) {
    fprintf(
        stderr, "GPUassert: %s %s %d\n", hipGetErrorString(error), file, line);
    if (abort) {
      exit(error);
    }
  }
}

__global__ void Memzero(float* x) {
  const int row = blockIdx.x;
  const int column = threadIdx.x;

  float* out = &x[row * blockDim.x + column];
  __stwt(out, 0.0f);
}

class L2CacheFlusher {
 public:
  L2CacheFlusher() {
    int device_id;
    CHECK_CUDA_ERROR(hipGetDevice(&device_id));

    CHECK_CUDA_ERROR(hipDeviceGetAttribute(
        &l2_cache_size_, hipDeviceAttributeL2CacheSize, device_id));
    std::cerr << "L2 cache size = " << l2_cache_size_ << std::endl;
    if (l2_cache_size_ <= 0) {
      std::cerr << "The L2 cache size is expected to be positive. Got "
                << l2_cache_size_ << std::endl;
      abort();
    }

    CHECK_CUDA_ERROR(hipMalloc(&buffer_, l2_cache_size_));
  }

  void Flush(hipStream_t stream) {
    constexpr int kWarpSize = 32;
    Memzero<<<
        l2_cache_size_ / sizeof(float) / kWarpSize,
        kWarpSize,
        0,
        stream>>>(reinterpret_cast<float*>(buffer_));
  }

  ~L2CacheFlusher() {
    if (buffer_ != nullptr) {
      CHECK_CUDA_ERROR(hipFree(buffer_));
    }
  }

 private:
  void* buffer_ = nullptr;
  int l2_cache_size_ = 0;
};

constexpr int kRows = 204800;
constexpr int kColumns = 512;

enum class CacheOp {
  Global,
  Streaming,
};

template <CacheOp cache_op>
__global__ void Axpy(const float alpha, const float* x, float* y) {
  const int row = blockIdx.x;
  const int column = threadIdx.x * 4;

  const float4* in =
      reinterpret_cast<const float4*>(&x[row * kColumns + column]);
  float4* out = reinterpret_cast<float4*>(&y[row * kColumns + column]);

  float4 vector;
  switch (cache_op) {
    case CacheOp::Global:
      vector = __ldcg(in);
      break;
    case CacheOp::Streaming:
      vector = __ldcs(in);
      break;
  }

  vector.x *= alpha;
  vector.y *= alpha;
  vector.z *= alpha;
  vector.w *= alpha;

  switch (cache_op) {
    case CacheOp::Global:
      __stcg(out, vector);
      break;
    case CacheOp::Streaming:
      __stcs(out, vector);
      break;
  }
}

int main(int argc, char* argv[]) {
  constexpr int kSize = kRows * kColumns;

  constexpr float alpha = 2.0f;
  std::vector<float> host_x(kSize);
  for (int i = 0; i < kSize; i++) {
    host_x[i] = i;
  }
  std::vector<float> host_y(kSize);

  L2CacheFlusher l2_cache_flusher;

  hipStream_t stream;
  CHECK_CUDA_ERROR(hipStreamCreate(&stream));

  float* device_x;
  float* device_y;
  CHECK_CUDA_ERROR(hipMallocAsync(&device_x, kSize * sizeof(float), stream));
  CHECK_CUDA_ERROR(hipMallocAsync(&device_y, kSize * sizeof(float), stream));

  CHECK_CUDA_ERROR(hipMemcpyAsync(
      device_x,
      host_x.data(),
      kSize * sizeof(float),
      hipMemcpyHostToDevice,
      stream));

  constexpr int kIterations = 3;
  for (int i = 0; i < kIterations; i++) {
    l2_cache_flusher.Flush(stream);
    Axpy<CacheOp::Global>
        <<<kRows, kColumns / 4, 0, stream>>>(alpha, device_x, device_y);
  }
  // Switching to __ldcs and __stcs slows the kernel down from 501us to 534us.
  for (int i = 0; i < kIterations; i++) {
    l2_cache_flusher.Flush(stream);
    Axpy<CacheOp::Streaming>
        <<<kRows, kColumns / 4, 0, stream>>>(alpha, device_x, device_y);
  }

  CHECK_CUDA_ERROR(hipMemcpyAsync(
      host_y.data(),
      device_y,
      kSize * sizeof(float),
      hipMemcpyDeviceToHost,
      stream));

  CHECK_CUDA_ERROR(hipFreeAsync(device_x, stream));
  CHECK_CUDA_ERROR(hipFreeAsync(device_y, stream));
  CHECK_CUDA_ERROR(hipStreamSynchronize(stream));
  CHECK_CUDA_ERROR(hipStreamDestroy(stream));

  // Compare actual and expected.
  for (int i = 0; i < kSize; ++i) {
    const float actual = host_y[i];
    const float expected = host_x[i] * alpha;
    if (fabs(actual - expected) > 1e-5) {
      std::cerr << "Mismatch at index " << i << ": expected = " << expected
                << ", actual = " << actual << std::endl;
      abort();
    }
  }

  return 0;
}
